﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

#define VECTOR_SIZE 1024 * 1024 * 512
#define THREADS_PER_BLOCK 256

#define CEIL(A, B) ((A) + (B) - 1) / (B)

#define BLOCK_NUM CEIL(VECTOR_SIZE, THREADS_PER_BLOCK)

#define CudaCheckError(error) \
	if (error != hipSuccess) \
		goto exit;

__global__ void add(int* a, int* b, int* result, size_t length)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < length)
		result[idx] = a[idx] + b[idx];
}

void fillArray(int* array, size_t size) 
{
	for (int i = 0; i < size; i++)
		array[i] = i;
}

bool checkResult(int* a, int* b, int* result, size_t size)
{
	for (int i = 0; i < size; i++)
	{
		if (a[i] + b[i] != result[i])
			return false;
	}
	return true;
}

int main(void)
{
	int* hA = NULL, * hB = NULL, * dA = NULL, * dB = NULL, * hResult = NULL, * dResult = NULL;

	CudaCheckError(hipHostAlloc(&hA, VECTOR_SIZE * sizeof(int), 0));
	CudaCheckError(hipHostAlloc(&hB, VECTOR_SIZE * sizeof(int), 0));
	CudaCheckError(hipHostAlloc(&hResult, VECTOR_SIZE * sizeof(int), 0));

	CudaCheckError(hipMalloc(&dA, VECTOR_SIZE * sizeof(int)));
	CudaCheckError(hipMalloc(&dB, VECTOR_SIZE * sizeof(int)));
	CudaCheckError(hipMalloc(&dResult, VECTOR_SIZE * sizeof(int)));

	fillArray(hA, VECTOR_SIZE);
	fillArray(hB, VECTOR_SIZE);

	CudaCheckError(hipMemcpy(dA, hA, VECTOR_SIZE * sizeof(int), hipMemcpyHostToDevice));
	CudaCheckError(hipMemcpy(dB, hB, VECTOR_SIZE * sizeof(int), hipMemcpyHostToDevice));

	add<<<BLOCK_NUM, THREADS_PER_BLOCK>>>(dA, dB, dResult, VECTOR_SIZE);
	CudaCheckError(hipGetLastError());
	CudaCheckError(hipDeviceSynchronize());

	CudaCheckError(hipMemcpy(hResult, dResult, VECTOR_SIZE * sizeof(int), hipMemcpyDeviceToHost));
	puts(checkResult(hA, hB, hResult, VECTOR_SIZE) ? "Result ok" : "Result not ok");

exit:
	if (hA) 
		hipHostFree(hA);
	if (hB) 
		hipHostFree(hB);
	if (hResult) 
		hipHostFree(hResult);

	if (dA)
		hipFree(dA);
	if (dB)
		hipFree(dB);
	if (dResult)
		hipFree(dResult);
	return 0;
}