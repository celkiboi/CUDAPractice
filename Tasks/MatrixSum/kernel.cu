#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define M 1025 * 512
#define N 1023
#define MATRIX_VECTOR_SIZE (M) * (N)

#define CEIL(A, B) (((A) + (B) - 1) / (B))

#define THREADS_PER_BLOCK_X 16
#define THREADS_PER_BLOCK_Y 16

#define BLOCKS_X CEIL(N, THREADS_PER_BLOCK_X)
#define BLOCKS_Y CEIL(M, THREADS_PER_BLOCK_Y)

#define CudaCheckError(error) \
	if (error != hipSuccess) \
		goto exit;

//#define DEBUG_PRINT
#ifdef DEBUG_PRINT
#define DEBUG_MATRIX_PRINT(MATRIX, DIM_M, DIM_N) \
	for (int i = 0; i < DIM_M; i++) \
	{ \
		for (int j = 0; j < DIM_N; j++) \
			printf("%d ", MATRIX[i * DIM_N + j]); \
		puts(""); \
	}
#else
#define DEBUG_MATRIX_PRINT(MATRIX, DIM_M, DIM_N)
#endif


__global__ void sum(int* matrixA, int* matrixB, size_t m, size_t n, int* result)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row * n + col;

	if (row >= m || col >= n)
		return;

	result[idx] = matrixA[idx] + matrixB[idx];
}

void fillMatrix(int* matrix, size_t matrixTotalSize)
{
	for (int i = 0; i < matrixTotalSize; i++)
		matrix[i] = i;
}

bool checkResult(int* matrixA, int* matrixB, int* result, size_t m, size_t n)
{
	for (int i = 0; i < m * n; i++)
	{
		if (matrixA[i] + matrixB[i] != result[i])
			return false;
	}

	return true;
}

int main(void)
{
	bool success = false;
	int* hA = NULL, * hB = NULL, * dA = NULL, * dB = NULL, * dResult = NULL, * hResult = NULL;

	CudaCheckError(hipHostAlloc(&hA, MATRIX_VECTOR_SIZE * sizeof(int), 0));
	CudaCheckError(hipHostAlloc(&hB, MATRIX_VECTOR_SIZE * sizeof(int), 0));
	CudaCheckError(hipHostAlloc(&hResult, MATRIX_VECTOR_SIZE * sizeof(int), 0));
	
	CudaCheckError(hipMalloc(&dA, MATRIX_VECTOR_SIZE * sizeof(int)));
	CudaCheckError(hipMalloc(&dB, MATRIX_VECTOR_SIZE * sizeof(int)));
	CudaCheckError(hipMalloc(&dResult, MATRIX_VECTOR_SIZE * sizeof(int)));

	fillMatrix(hA, MATRIX_VECTOR_SIZE);
	fillMatrix(hB, MATRIX_VECTOR_SIZE);

	CudaCheckError(hipMemcpy(dA, hA, MATRIX_VECTOR_SIZE * sizeof(int), hipMemcpyHostToDevice));
	CudaCheckError(hipMemcpy(dB, hB, MATRIX_VECTOR_SIZE * sizeof(int), hipMemcpyHostToDevice));

	dim3 threadsPerBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
	dim3 blocksPerGrid(BLOCKS_X, BLOCKS_Y);
	sum<<<blocksPerGrid, threadsPerBlock>>>(dA, dB, M, N, dResult);
	CudaCheckError(hipGetLastError());
	CudaCheckError(hipDeviceSynchronize());

	CudaCheckError(hipMemcpy(hResult, dResult, MATRIX_VECTOR_SIZE * sizeof(int), hipMemcpyDeviceToHost));
	DEBUG_MATRIX_PRINT(hResult, M, N);
	puts(checkResult(hA, hB, hResult, M, N) ? "Result ok" : "Result not ok");
	
	success = true;
exit:
	if (hA)
		hipHostFree(hA);
	if (hB)
		hipHostFree(hB);
	if (hResult)
		hipHostFree(hResult);

	if (dA)
		hipFree(dA);
	if (dB)
		hipFree(dB);
	if (dResult)
		hipFree(dResult);

	if (!success)
		printf("Cuda error: %s", hipGetErrorString(hipGetLastError()));

	return 0;
}